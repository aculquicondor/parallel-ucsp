#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <chrono>

using namespace std;

__global__ void matrix_product_kernel(float *P, float *M, float *N,
                                      size_t width) {
  int i = threadIdx.x, j = threadIdx.y;
  float acc = 0;
  for (int k = 0; k < width; ++k)
    acc += M[i*width+k] * N[k*width+j];
  P[i*width+j] = acc;
}

void matrix_product_serial(float *P, float *M, float *N, size_t width) {
#pragma omp parallel for
  for (size_t i = 0; i < width; ++i)
    for (size_t j = 0; j < width; ++j) {
      float acc = 0;
      for (size_t k = 0; k < width; ++k)
        acc += M[i*width+k] * N[k*width+j];
      P[i*width+j] = acc;
    }
}

void matrix_product(float *P, float *M, float *N, size_t width) {
  float *Pd, *Md, *Nd;
  size_t dsize = width * width * sizeof(float);
  hipMalloc((void **)&Md, dsize);
  hipMemcpy(Md, M, dsize, hipMemcpyHostToDevice);
  hipMalloc((void **)&Nd, dsize);
  hipMemcpy(Nd, N, dsize, hipMemcpyHostToDevice);
  hipMalloc((void **)&Pd, dsize);

  dim3 dim_block(width, width);
  dim3 dim_grid(1, 1);
  matrix_product_kernel<<<dim_grid, dim_block>>>(Pd, Md, Nd, width);

  hipMemcpy(P, Pd, dsize, hipMemcpyDeviceToHost);

  hipFree(Pd);
  hipFree(Md);
  hipFree(Nd);
}

void print(float *M, size_t width) {
  for (size_t i = 0; i < width; ++i) {
    for (size_t j = 0; j < width; ++j)
      printf("%.2f ", M[i*width+j]);
    puts("");
  }
}

int main(int argc, char *argv[]) {
  size_t width;

#ifdef NOIO
  width = 512;
#else
  scanf("%ld", &width);
#endif

  float *M = new float[width*width],
        *N = new float[width*width],
        *P = new float[width*width];
  for (int i = 0; i < width * width; ++i) {
    M[i] = ((float)rand() / RAND_MAX - 0.5) * 2;
    N[i] = ((float)rand() / RAND_MAX - 0.5) * 2;
  }

  chrono::time_point<chrono::system_clock> start, end;
  int ms;

  start = chrono::system_clock::now();
  matrix_product(P, M, N, width);
  end = chrono::system_clock::now();
  ms = chrono::duration_cast<chrono::milliseconds>(end-start).count();
  printf("GPU: %d\n", ms);

#ifndef NOIO
  print(M, width);
  puts("-------");
  print(N, width);
  puts("-------");
  print(P, width);
#endif

  start = chrono::system_clock::now();
  matrix_product_serial(P, M, N, width);
  end = chrono::system_clock::now();
  ms = chrono::duration_cast<chrono::milliseconds>(end-start).count();
  printf("CPU: %d\n", ms);

#ifndef NOIO
  print(M, width);
  puts("-------");
  print(N, width);
  puts("-------");
  print(P, width);
#endif

  delete[] M;
  delete[] N;
  delete[] P;
  return 0;
}

