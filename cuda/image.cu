#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <chrono>
#define cimg_display 0
#include <hip/hip_runtime.h>
#include <CImg.h>

using namespace std;
using namespace cimg_library;

const int block_row = 32;

__global__ void img_filter_kernel(unsigned char *input, unsigned char *output,
                                  size_t r, size_t c) {
  int x = blockIdx.x * block_row + threadIdx.x,
      y = blockIdx.y * block_row + threadIdx.y,
      z = blockIdx.z;
  int base = r * c * z;
  int sum = 0, count = 0;
  for (int i = y - 1; i <= y + 1; ++i)
    if (i >= 0 && i < r)
      for (int j = x - 1; j <= x + 1; ++j)
        if (j >= 0 && j < c) {
          ++count;
          sum += input[base + c * i + j];
        }
  if (count)
    output[base + c * y + x] = sum / count;
}

typedef CImg<unsigned char> Image;

void img_filter(Image &img) {
  unsigned char *input_d, *output_d;
  size_t data_size = img.size() * sizeof(unsigned char);
  hipMalloc((void **)&input_d, data_size);
  hipMalloc((void **)&output_d, data_size);
  hipMemcpy(input_d, img.data(), data_size, hipMemcpyHostToDevice);
  size_t r = img.height(), c = img.width();
  dim3 grid_dim((c+block_row-1)/block_row, (r+block_row-1)/block_row,
                img.spectrum()), block_dim(block_row, block_row);
  img_filter_kernel<<<grid_dim, block_dim>>>(input_d, output_d, r, c);
  hipMemcpy(img.data(), output_d, data_size, hipMemcpyDeviceToHost);
  hipFree(input_d);
  hipFree(output_d);
}

int main(int argc, char *argv[]) {
  Image img("lena.jpg");

  chrono::time_point<chrono::system_clock> start, end;
  int ms;
  start = chrono::system_clock::now();

  img_filter(img);

  end = chrono::system_clock::now();
  ms = chrono::duration_cast<chrono::milliseconds>(end-start).count();
  printf("GPU: %d\n", ms);

  img.save("lena_filter.jpg");

  return 0;
}

